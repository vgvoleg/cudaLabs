#include "hip/hip_runtime.h"
#include "blas_cuda.h"
#include <hip/hip_runtime_api.h>

__global__ void saxpy_GPU(int n, float a, float *x, int incx, float *y, int incy) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if ( (i*incy < n) && (i*incx < n) ) {
    y[i*incy] += a*x[i*incx];
  }
}

__global__ void daxpy_GPU(int n, double a, double *x, int incx, double *y, int incy) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if ( (i*incy < n) && (i*incx < n) ) {
    y[i*incy] += a*x[i*incx];
  }
}

void saxpy_gpu(int n, float a, float *x, int incx, float *y, int incy) {
  float *x_gpu, *y_gpu;
  hipMalloc((void**)&x_gpu, n*sizeof(float));
  hipMalloc((void**)&y_gpu, n*sizeof(float));

  hipMemcpy(x_gpu, x, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y_gpu, y, n*sizeof(float), hipMemcpyHostToDevice);

  int num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  saxpy_GPU <<< num_blocks, BLOCK_SIZE >>> (n, a, x_gpu, incx, y_gpu, incy);

  hipMemcpy(y, y_gpu, n*sizeof(float), hipMemcpyDeviceToHost);
  
  hipFree(x_gpu);
  hipFree(y_gpu);
}
